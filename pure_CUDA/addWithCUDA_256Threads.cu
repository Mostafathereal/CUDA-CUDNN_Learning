
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float*y){

    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i+= stride){
        y[i] = y[i] + x[i];
    }
}

int validate(float *y){
    bool flag = true;
    for(int i = 0; i < 1<<20; i++){
        flag &= (y[i] == 3.0);
    }
    return flag;
}

int main(void){

    int N = 1<<20;
    
    float *x;
    float *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for(int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add<<<1, 256>>>(N, x, y);

    // wait for GPU to finihs before accessing on host (wait up cpu)
    hipDeviceSynchronize();

    // std::cout << validate(y) << "\n";

    hipFree(x);
    hipFree(y);

    

    return 0;



}