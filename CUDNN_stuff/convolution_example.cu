
#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
 
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

cv::Mat ldimag(const char* image_path){
    cv::Mat image = cv::imread(image_path);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

int main(void){
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    cv::Mat image = ldimag("conure.jpg");

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                            HIPDNN_TENSOR_NCHW, 
                                            HIPDNN_DATA_FLOAT, 
                                            1, 
                                            3,
                                            image.rows,
                                            image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            1, 
                                            3, 
                                            image.rows,
                                            image.cols));

    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_TENSOR_NCHW,
                                            3,
                                            3,
                                            3,
                                            3));

    hipdnnConvolutionDescriptor_t conv_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_descriptor,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            CUDNN_CROSS_CORRILATION,
                                            HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t conv_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            filter_descriptor,
                                            conv_descriptor,
                                            output_descriptor,
                                            CUDNN_CONVOLUTION_FWD_PREFER_FAST,
                                            0,
                                            &conv_alg));

    size_t ws_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                            input_descriptor, 
                                            filter_descriptor, 
                                            conv_descriptor,
                                            output_descriptor,
                                            conv_alg,
                                            &ws_bytes));
                                        
    std::cout << "WS size: " << (ws_bytes / (1<<20)) << "- MB" << std::endl;


    

    



    return 0;
}