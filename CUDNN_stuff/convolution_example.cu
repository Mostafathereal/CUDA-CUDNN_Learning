
#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
 
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

void save_image(const char* output_filename,
    float* buffer,
    int height,
    int width) {
cv::Mat output_image(height, width, CV_32FC3, buffer);
// Make negative values zero.
cv::threshold(output_image,
    output_image,
    /*threshold=*/0,
    /*maxval=*/0,
    cv::THRESH_TOZERO);
cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
output_image.convertTo(output_image, CV_8UC3);
cv::imwrite(output_filename, output_image);
std::cerr << "Wrote output to " << output_filename << std::endl;
}

cv::Mat ldimag(const char* image_path){
    cv::Mat image = cv::imread(image_path);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

int main(void){
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    cv::Mat image = ldimag("conure.jpg");

    // define descriptors for input/output tensors, and for filter tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                            HIPDNN_TENSOR_NHWC, 
                                            HIPDNN_DATA_FLOAT, 
                                            1, 
                                            3,
                                            image.rows,
                                            image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                            HIPDNN_TENSOR_NHWC,
                                            HIPDNN_DATA_FLOAT,
                                            1, 
                                            3, 
                                            image.rows,
                                            image.cols));

    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_TENSOR_NCHW,
                                            3,
                                            3,
                                            3,
                                            3));
    
    // define type of convolution (same padding, stride of 1 (for h and w), and no dilation (=1))                                         
    hipdnnConvolutionDescriptor_t conv_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_descriptor))
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            HIPDNN_CROSS_CORRELATION,
                                            HIPDNN_DATA_FLOAT));

    // define the convolution algorithm, using the type of conv described above                                        
    hipdnnConvolutionFwdAlgo_t conv_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            filter_descriptor,
                                            conv_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &conv_alg));
    // allocate buffer memeory for system to execute algorithm, first find size of workspace required (depends on conv_alg)
    size_t ws_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                            input_descriptor, 
                                            filter_descriptor, 
                                            conv_descriptor,
                                            output_descriptor,
                                            conv_alg,
                                            &ws_bytes));
                                        
    std::cout << "WS size: " << (ws_bytes / (1<<20)) << "- MB" << std::endl;

    // size of input img in bytes
    int imsize = image.rows * image.cols * 3 * sizeof(float);

    void* d_ws = nullptr;
    hipMalloc(&d_ws, ws_bytes);

    // allocate h * w * channels * float size to the input tensor
    float* d_input = nullptr;
    hipMalloc(&d_input, imsize);

    // since it is a `same` convolution, use same amount of memory for output 
    float* d_output = nullptr;
    hipMalloc(&d_output, imsize);

    hipMemcpy(d_input, image.ptr<float>(0), imsize, hipMemcpyHostToDevice);
    hipMemset(d_output, 0, imsize);

    // 3x3 size of kernel, x3 to match #channels of input image, (resulting in a 1 channel out-img)
    // again x3 to result in the same number of channels in output as input
    float kernel_temp[3][3][3][3] = {
    {{{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},     
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},      
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}}},     
    {{{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},     
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},     
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}}},     
    {{{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},     
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}},     
    {{-2, 0, 2},
    {-5, 0, 5},
    {-2, 0, 2}}}};

    float* d_kernel = nullptr;
    hipMalloc(&d_kernel, sizeof(kernel_temp));
    hipMemcpy(d_kernel, kernel_temp, sizeof(kernel_temp), hipMemcpyHostToDevice);

    //performing the convolution
    float alpha, beta; 
    alpha = 1;
    beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn, 
                                        &alpha,
                                        input_descriptor,
                                        d_input,
                                        filter_descriptor,
                                        d_kernel, 
                                        conv_descriptor,
                                        conv_alg,
                                        d_ws,
                                        ws_bytes,
                                        &beta,
                                        output_descriptor,
                                        d_output));

    std::cout << "\n\n hehe after conv \n\n\n";


    float* h_output = new float[imsize];

    std::cout << "\n\n hehe after conv 1\n\n\n";

    hipMemcpy(h_output, d_output, imsize, hipMemcpyDeviceToHost);

    std::cout << "\n\n hehe after conv 2\n\n\n";

    save_image("convoluted_conure(out).png", h_output, image.rows, image.cols);

    std::cout << "\n\n hehe after conv 3\n\n\n" << imsize << "\n\n";


    delete[] h_output;
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_ws);
    hipFree(d_output);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);

    hipdnnDestroy(cudnn);

    return 0;
}