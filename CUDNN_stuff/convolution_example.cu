
#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
 
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

cv::Mat ldimag(const char* image_path){
    cv::Mat image = cv::imread(image_path);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

int main(void){
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    cv::Mat image = ldimag("conure.jpg");

    // define descriptors for input/output tensors, and for filter tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                            HIPDNN_TENSOR_NCHW, 
                                            HIPDNN_DATA_FLOAT, 
                                            1, 
                                            3,
                                            image.rows,
                                            image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            1, 
                                            3, 
                                            image.rows,
                                            image.cols));

    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_TENSOR_NCHW,
                                            3,
                                            3,
                                            3,
                                            3));
    
    // define type of convolution (same padding, stride of 1 (for h and w), and no dilation (=1))                                         
    hipdnnConvolutionDescriptor_t conv_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_descriptor))
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            HIPDNN_CROSS_CORRELATION,
                                            HIPDNN_DATA_FLOAT));

    // define the convolution algorithm, using the type of conv described above                                        
    hipdnnConvolutionFwdAlgo_t conv_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            filter_descriptor,
                                            conv_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &conv_alg));
    // allocate buffer memeory for system to execute algorithm, first find size of workspace required (depends on conv_alg)
    size_t ws_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                            input_descriptor, 
                                            filter_descriptor, 
                                            conv_descriptor,
                                            output_descriptor,
                                            conv_alg,
                                            &ws_bytes));
                                        
    std::cout << "WS size: " << (ws_bytes / (1<<20)) << "- MB" << std::endl;

    

    

    



    return 0;
}