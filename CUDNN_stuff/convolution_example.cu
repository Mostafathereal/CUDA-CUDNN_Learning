
#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
 
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

cv::Mat ldimag(const char* image_path){
    cv::Mat image = cv::imread(image_path);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

int main(void){
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    cv::Mat image = ldimag("conure.jpg");

    // define descriptors for input/output tensors, and for filter tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                            HIPDNN_TENSOR_NCHW, 
                                            HIPDNN_DATA_FLOAT, 
                                            1, 
                                            3,
                                            image.rows,
                                            image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, 
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            1, 
                                            3, 
                                            image.rows,
                                            image.cols));

    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor, 
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_TENSOR_NCHW,
                                            3,
                                            3,
                                            3,
                                            3));
    
    // define type of convolution (same padding, stride of 1 (for h and w), and no dilation (=1))                                         
    hipdnnConvolutionDescriptor_t conv_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_descriptor))
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            1,
                                            HIPDNN_CROSS_CORRELATION,
                                            HIPDNN_DATA_FLOAT));

    // define the convolution algorithm, using the type of conv described above                                        
    hipdnnConvolutionFwdAlgo_t conv_alg;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            filter_descriptor,
                                            conv_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &conv_alg));
    // allocate buffer memeory for system to execute algorithm, first find size of workspace required (depends on conv_alg)
    size_t ws_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                            input_descriptor, 
                                            filter_descriptor, 
                                            conv_descriptor,
                                            output_descriptor,
                                            conv_alg,
                                            &ws_bytes));
                                        
    std::cout << "WS size: " << (ws_bytes / (1<<20)) << "- MB" << std::endl;

    // size of input img in bytes
    imsize = image.rows * image.cols * 3 * sizeof(float)

    void* d_ws = nullptr;
    hipMalloc(&d_ws, ws_bytes);

    // allocate h * w * channels * float size to the input tensor
    float* d_input = nullptr;
    hipMalloc(&d_input, imsize);

    // since it is a `same` convolution, use same amount of memory for output 
    float* d_output = nullptr;
    hipMalloc(&d_input, imsize);

    hipMemcpy(d_input, image.ptr<float>0, imsize, hipMemcpyHostToDevice);
    hipMemset(d_output, 0, imsize);

    // 3x3 size of kernel, x3 to match #channels of input image, (resulting in a 1 channel out-img)
    // again x3 to result in the same number of channels in output as input
    float kernel_temp[3][3][[3][3] = 
    {{{{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}}},     
     {{{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}}},     
     {{{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}},     
     {{-2, 0, 2},
     {-5, 0, 5},
     {-2, 0, 2}}}}

    float* d_kernel = nullptr;
    hipMalloc(d_kernel, sizeof(kernel_temp))
    hipMemcpy(d_kernel, kernel_temp, sizeof(kernel_temp), hipMemcpyHostToDevice)

    //performing the convolution
    float alpha, beta = 1, 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn, 
                                        &alpha,
                                        input_descriptor,
                                        d_input,
                                        filter_descriptor,
                                        d_kernel, 
                                        conv_descriptor,
                                        conv_alg,
                                        d_ws,
                                        &beta,
                                        output_descriptor,
                                        d_output));


    // float* h_output = new float[imsize];
    // hipMemcpy()
    


    



    return 0;
}