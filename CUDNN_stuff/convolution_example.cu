
#include <hipDNN.h>
#include <iostream>
#include <opencv2/opencv.hpp>
 
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

cv::Mat ldimag(const char* image_path){
    cv::Mat image = cv::imread(image_path);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

int main(void){
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    cv::Mat image = ldimag("conure.jpg");

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, 
                                            HIPDNN_TENSOR_NCHW, 
                                            HIPDNN_DATA_FLOAT, 
                                            1, 
                                            3,
                                            image.rows,
                                            image.cols));

    



    return 0;
}